#include <hip/hip_runtime.h>

#include <cmath>
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <assert.h>
#include <hipfft/hipfft.h>

#define PI (3.141592653589793238462643383279502884197169399375105820974944592307816406286208998628034825342117067982148086513282306647093844609550582231725359408128481)
#define TPB (16)
#define NUM_RUNS (5)

#if 1
typedef float dtype;
typedef hipfftReal dtypeReal;
typedef hipfftComplex dtypeComplex;
#define epsilon (5e-1) //relative error
#else
typedef double dtype;
typedef cufftDoubleReal dtypeReal;
typedef cufftDoubleComplex dtypeComplex;
#define epsilon (1e-2) //relative error
#endif

#define checkCUDA(status)                       \
    {                                           \
        if (status != hipSuccess)              \
        {                                       \
            printf("CUDA Runtime Error: %s\n",  \
                   hipGetErrorString(status)); \
            assert(status == hipSuccess);      \
        }                                       \
    }

typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void)
{
    using namespace std::chrono;
    return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void)
{
    using namespace std::chrono;
    return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}

hr_clock_rep timer_start, timer_stop;

/// Return true if a number is power of 2
template <typename T = unsigned>
inline bool isPowerOf2(T val)
{
    return val && (val & (val - 1)) == 0;
}

inline __device__ int INDEX(const int hid, const int wid, const int N)
{
    return (hid * N + wid);
}

template <typename T>
__global__ void reorderInput(const T *x, T *y, const int M, const int N)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond)
        {
        case 0:
            index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
            break;
        case 1:
            index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
            break;
        case 2:
            index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
            break;
        case 3:
            index = INDEX(hid << 1, wid << 1, N);
            break;
        default:
            assert(0);
            break;
        }
        y[INDEX(hid, wid, N)] = x[index];
    }
}

inline __device__ hipfftDoubleComplex complexMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

inline __device__ hipfftComplex complexMul(const hipfftComplex &x, const hipfftComplex &y)
{
    hipfftComplex res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

inline __device__ hipfftDoubleReal RealPartOfMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    return x.x * y.x - x.y * y.y;
}

inline __device__ hipfftReal RealPartOfMul(const hipfftComplex &x, const hipfftComplex &y)
{
    return x.x * y.x - x.y * y.y;
}

inline __device__ hipfftDoubleComplex complexAdd(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

inline __device__ hipfftComplex complexAdd(const hipfftComplex &x, const hipfftComplex &y)
{
    hipfftComplex res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

inline __device__ hipfftDoubleComplex complexConj(const hipfftDoubleComplex &x)
{
    hipfftDoubleComplex res;
    res.x = x.x;
    res.y = -1 * x.y;
    return res;
}

inline __device__ hipfftComplex complexConj(const hipfftComplex &x)
{
    hipfftComplex res;
    res.x = x.x;
    res.y = -1 * x.y;
    return res;
}

__global__ void precomputeExpk(hipfftDoubleComplex *expkM, hipfftDoubleComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftDoubleComplex W_h_4M = make_double2(cos(PI * hid / (2 * M)), -1 * sin(PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid < N)
    {
        int wid = tid;
        hipfftDoubleComplex W_w_4N = make_double2(cos(PI * wid / (2 * N)), -1 * sin(PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

__global__ void precomputeExpk(hipfftComplex *expkM, hipfftComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftComplex W_h_4M = make_float2(cos(PI * hid / (2 * M)), -1 * sin(PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid < N)
    {
        int wid = tid;
        hipfftComplex W_w_4N = make_float2(cos(PI * wid / (2 * N)), -1 * sin(PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

template <typename T, typename TComplex>
__global__ __launch_bounds__(512, 2)
void computeMulExpk(const TComplex *V, T *y, const int M, const int N,
                               const TComplex *__restrict__ expkM,
                               const TComplex *__restrict__ expkN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        if (hid == 0)
        {
            TComplex tmp;
            if (wid <= N / 2)
            {
                tmp = V[wid];
            }
            else
            {
                tmp = complexConj(V[N - wid]);
            }
            // tmp = complexMul(expkM[0], tmp); // expkM[0] = 1
            y[wid] = RealPartOfMul(expkN[wid], tmp) * 4. / (M * N);
        }
        else
        {
            TComplex tmp1, tmp2, tmp;
            if (wid <= N / 2)
            {
                tmp1 = V[INDEX(hid, wid, N / 2 + 1)];
                tmp2 = V[INDEX(M - hid, wid, N / 2 + 1)];
                tmp.x = expkM[hid].x * (tmp1.x + tmp2.x) + expkM[hid].y * (tmp2.y - tmp1.y);
                tmp.y = expkM[hid].x * (tmp1.y + tmp2.y) + expkM[hid].y * (tmp1.x - tmp2.x);
            }
            else
            {
                tmp1 = V[INDEX(M - hid, N - wid, N / 2 + 1)];
                tmp2 = V[INDEX(hid, N - wid, N / 2 + 1)];
                tmp.x = expkM[hid].x * (tmp1.x + tmp2.x) + expkM[hid].y * (tmp1.y - tmp2.y);
                tmp.y = expkM[hid].y * (tmp1.x - tmp2.x) - expkM[hid].x * (tmp1.y + tmp2.y);
            }
            y[INDEX(hid, wid, N)] = RealPartOfMul(expkN[wid], tmp) * 2. / (M * N);
        }
    }
}

template <typename T>
void fft2D(T *d_x, hipfftDoubleComplex *d_y, const int M, const int N)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, M, N, HIPFFT_D2Z);
    hipfftExecD2Z(plan, (hipfftDoubleReal*)d_x, d_y);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
}

template <typename T>
void fft2D(T *d_x, hipfftComplex *d_y, const int M, const int N)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, M, N, HIPFFT_R2C);
    hipfftExecR2C(plan, (hipfftReal*)d_x, d_y);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
}

template <typename T, typename TDouble=hipfftDoubleReal, typename TComplex=hipfftDoubleComplex>
void dct_2d_fft(const T *h_x, T *h_y, const int M, const int N)
{
    T *d_x;
    T *d_y;
    TComplex *scratch;
    TComplex *expkM, *expkN;

    if (!isPowerOf2<int>(N) || !isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = M * N * sizeof(T);
    checkCUDA(hipMalloc((void **)&d_x, size));
    checkCUDA(hipMalloc((void **)&expkM, M * sizeof(TComplex)));
    checkCUDA(hipMalloc((void **)&expkN, N * sizeof(TComplex)));

    checkCUDA(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    precomputeExpk<<<(std::max(M, N) + 1023) / 1024, 1024>>>(expkM, expkN, M, N);
    hipDeviceSynchronize();

    timer_start = get_globaltime();
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&scratch, M*(N/2+1)*sizeof(TComplex));

    reorderInput<T><<<gridSize, blockSize>>>(d_x, d_y, M, N);
    hipDeviceSynchronize();

    fft2D(d_y, scratch, M, N);

    computeMulExpk<T, TComplex><<<gridSize, blockSize>>>(scratch, d_y, M, N, expkM, expkN);
    hipDeviceSynchronize();
    timer_stop = get_globaltime();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(scratch);
    hipFree(expkM);
    hipFree(expkN);
}

template <typename T>
int validate_fft(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag = (std::abs(result_cuda[(i * N + j) << 1] - result_gt[(i * N + j) << 1]) / std::abs(result_gt[(i * N + j) << 1])) < epsilon;
            if (flag == 0)
            {
                // printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i*N+j], i, j, result_gt[i*N+j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
int validate2D(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j]) / std::abs(result_gt[i * N + j])) < epsilon;
            if (flag == 0)
            {
                // printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i*N+j], i, j, result_gt[i*N+j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
T **allocateMatrix(int M, int N)
{
    T **data;
    data = new T *[M];
    for (int i = 0; i < M; i++)
    {
        data[i] = new T[N];
    }
    return data;
}

template <typename T>
void destroyMatrix(T **&data, int M)
{
    for (int i = 0; i < M; i++)
    {
        delete[] data[i];
    }
    delete[] data;
}

template <typename T>
void load_data(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N];
    while (input_file2 >> val)
    {
        result[i] = val;
        i++;
    }
    printf("[I] data load done.\n");
}

template <typename T>
void load_data_fft(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d_fft.dat", std::ios_base::in);

    int i = 0;
    T val, imag;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d_fft.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N * 2];
    while (input_file2 >> val >> imag)
    {
        result[i] = val;
        result[i + 1] = imag;
        i += 2;
    }
    printf("[I] data load done.\n");
}

int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    // load_data_fft<dtype>(h_x, h_gt, M, N);
    h_y = new dtype[M * N];
    // h_y = new dtype[M * N * 2];

    double total_time = 0;
    for (int i = 0; i < NUM_RUNS; ++i)
    {
        dct_2d_fft<dtype, dtypeReal, dtypeComplex>(h_x, h_y, M, N);
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        // int flag = validate_fft<dtype>(h_y, h_gt, M ,N);
        if (!flag)
        {
            printf("[I] Error! Results are incorrect.\n", flag);
            for (int i = 0; i < 4; ++i)
            {
                printf("index: %d, result: %f, GT: %f, scale: %f\n", i, h_y[i], h_gt[i], h_y[i] / h_gt[i]);
            }
        }
        printf("[D] dct 2D takes %g ms\n", (timer_stop - timer_start) * get_timer_period());
        total_time += i > 0 ? (timer_stop - timer_start) * get_timer_period() : 0;
    }

    printf("[D] dct 2D (%d * %d) takes average %g ms\n", M, N, total_time / (NUM_RUNS - 1));

    delete[] h_x;
    delete[] h_y;
    delete[] h_gt;

    return 0;
}
