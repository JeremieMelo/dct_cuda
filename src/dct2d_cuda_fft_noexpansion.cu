#include <hip/hip_runtime.h>

#include <cmath>
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <assert.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>

#define PI (3.141592653589793238462643383279502884197169399375105820974944592307816406286208998628034825342117067982148086513282306647093844609550582231725359408128481)
#define TPB (32)
#define epsilon (1e-2) //relative error
#define NUM_RUNS (5)

#define checkCUDA(status)                       \
    {                                           \
        if (status != hipSuccess)              \
        {                                       \
            printf("CUDA Runtime Error: %s\n",  \
                   hipGetErrorString(status)); \
            assert(status == hipSuccess);      \
        }                                       \
    }

typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void)
{
    using namespace std::chrono;
    return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void)
{
    using namespace std::chrono;
    return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}

hr_clock_rep timer_start, timer_stop;

/// Return true if a number is power of 2
template <typename T = unsigned>
inline bool isPowerOf2(T val)
{
    return val && (val & (val - 1)) == 0;
}

inline __device__ int INDEX(const int hid, const int wid, const int N)
{
    return (hid * N + wid);
}

template <typename T>
__global__ void reorderInput(const T *x, T *y, const int M, const int N)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond)
        {
        case 0:
            index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
            break;
        case 1:
            index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
            break;
        case 2:
            index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
            break;
        case 3:
            index = INDEX(hid << 1, wid << 1, N);
            break;
        default:
            assert(0);
            break;
        }
        y[INDEX(hid, wid, N)] = x[index];
    }
}

inline __device__ hipfftDoubleComplex complexMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

inline __device__ hipfftDoubleReal RealPartOfMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    return x.x * y.x - x.y * y.y;
}

inline __device__ hipfftDoubleComplex complexAdd(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

inline __device__ hipfftDoubleComplex complexAverage(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = (x.x + y.x) / 2.;
    res.y = (x.y + y.y) / 2.;
    return res;
}

inline __device__ hipfftDoubleComplex complexConj(const hipfftDoubleComplex &x)
{
    hipfftDoubleComplex res;
    res.x = x.x;
    res.y = -1 * x.y;
    return res;
}

__global__ void precomputeExpk(hipfftDoubleComplex *expkM, hipfftDoubleComplex *expkN, hipfftDoubleComplex *expkMconj, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftDoubleComplex W_h_4M = make_double2(cos(PI * hid / (2 * M)), -1 * sin(PI * hid / (M * 2)));
        hipfftDoubleComplex W_h_4M_conj = make_double2(W_h_4M.x, -1 * W_h_4M.y);
        expkM[hid] = W_h_4M;
        expkMconj[hid] = W_h_4M_conj;
    }
    if (tid < N)
    {
        int wid = tid;
        hipfftDoubleComplex W_w_4N = make_double2(cos(PI * wid / (2 * N)), -1 * sin(PI * wid / (N * 2)));
        // cufftDoubleComplex W_w_4N_conj = make_double2(W_w_4N.x, -1 * W_w_4N.y);
        expkN[wid] = W_w_4N;
        // expkNconj[wid] = W_w_4N_conj;
    }
}

template <typename T>
__global__ void computeMulExpk(const hipfftDoubleComplex *V, T *y, const int M, const int N,
                               const hipfftDoubleComplex *__restrict__ expkM,
                               const hipfftDoubleComplex *__restrict__ expkN,
                               const hipfftDoubleComplex *__restrict__ expkMconj)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        if (hid == 0)
        {
            hipfftDoubleComplex tmp;
            if (wid <= N / 2)
            {
                tmp = V[wid];
            }
            else
            {
                tmp = complexConj(V[N - wid]);
            }
            // tmp = complexMul(expkM[0], tmp); // expkM[0] = 1
            y[wid] = RealPartOfMul(expkN[wid], tmp) * 4. / (M * N);
        }
        else
        {
            hipfftDoubleComplex tmp1, tmp2;
            if (wid <= N / 2)
            {
                tmp1 = V[INDEX(hid, wid, N / 2 + 1)];
                tmp2 = V[INDEX(M - hid, wid, N / 2 + 1)];
            }
            else
            {
                tmp1 = complexConj(V[INDEX(M - hid, N - wid, N / 2 + 1)]);
                tmp2 = complexConj(V[INDEX(hid, N - wid, N / 2 + 1)]);
            }

            tmp1 = complexMul(expkM[hid], tmp1);
            tmp2 = complexMul(expkMconj[hid], tmp2);
            tmp1 = complexAverage(tmp1, tmp2);
            y[INDEX(hid, wid, N)] = RealPartOfMul(expkN[wid], tmp1) * 4. / (M * N);
        }
    }
}

template <typename T>
void dct_1d_z2z(hipfftDoubleComplex *d_x,
                hipfftDoubleComplex *d_y,
                const int M,
                const int N)
{
    hipfftHandle plan;
    int n[1] = {N};
    int BATCH = M / 2;

    hipfftPlanMany(&plan, 1, n,
                  NULL, 1, N,
                  NULL, 1, N,
                  HIPFFT_Z2Z, BATCH);
    hipfftExecZ2Z(plan, d_x, d_y, HIPFFT_FORWARD);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
}

template <typename T>
void dct_2d_d2z(hipfftDoubleReal *d_x,
                hipfftDoubleComplex *d_y,
                const int M,
                const int N)
{
    hipfftHandle plan;
    hipfftPlan2d(&plan, M, N, HIPFFT_D2Z);
    hipfftExecD2Z(plan, d_x, d_y);
    hipDeviceSynchronize();
    hipfftDestroy(plan);
}

template <typename T>
void dct_2d_fft(const T *h_x, T *h_y, const int M, const int N)
{
    hipfftDoubleReal *d_x;
    T *d_y;
    hipfftDoubleComplex *scratch;
    hipfftDoubleComplex *expkM, *expkN, *expkMconj;

    if (!isPowerOf2<int>(N) || !isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = M * N * sizeof(T);
    checkCUDA(hipMalloc((void **)&d_x, size));
    checkCUDA(hipMalloc((void **)&expkM, M * sizeof(hipfftDoubleComplex)));
    checkCUDA(hipMalloc((void **)&expkN, N * sizeof(hipfftDoubleComplex)));
    checkCUDA(hipMalloc((void **)&expkMconj, M * sizeof(hipfftDoubleComplex)));

    checkCUDA(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));
    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    precomputeExpk<<<(std::max(M, N) + 1023) / 1024, 1024>>>(expkM, expkN, expkMconj, M, N);
    hipDeviceSynchronize();

    timer_start = get_globaltime();
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&scratch, size * 2);

    reorderInput<T><<<gridSize, blockSize>>>(d_x, d_y, M, N);
    hipDeviceSynchronize();

    dct_2d_d2z<T>(d_y, scratch, M, N);

    computeMulExpk<T><<<gridSize, blockSize>>>(scratch, d_y, M, N, expkM, expkN, expkMconj);
    hipDeviceSynchronize();
    timer_stop = get_globaltime();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(scratch);
    hipFree(expkM);
    hipFree(expkN);
    hipFree(expkMconj);
}

template <typename T>
int validate_fft(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag = (std::abs(result_cuda[(i * N + j) << 1] - result_gt[(i * N + j) << 1]) / std::abs(result_gt[(i * N + j) << 1])) < epsilon;
            if (flag == 0)
            {
                // printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i*N+j], i, j, result_gt[i*N+j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
int validate2D(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j]) / std::abs(result_gt[i * N + j])) < epsilon;
            if (flag == 0)
            {
                // printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i*N+j], i, j, result_gt[i*N+j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
T **allocateMatrix(int M, int N)
{
    T **data;
    data = new T *[M];
    for (int i = 0; i < M; i++)
    {
        data[i] = new T[N];
    }
    return data;
}

template <typename T>
void destroyMatrix(T **&data, int M)
{
    for (int i = 0; i < M; i++)
    {
        delete[] data[i];
    }
    delete[] data;
}

template <typename T>
void load_data(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N];
    while (input_file2 >> val)
    {
        result[i] = val;
        i++;
    }
    printf("[I] data load done.\n");
}

template <typename T>
void load_data_fft(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d_fft.dat", std::ios_base::in);

    int i = 0;
    T val, imag;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d_fft.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N * 2];
    while (input_file2 >> val >> imag)
    {
        result[i] = val;
        result[i + 1] = imag;
        i += 2;
    }
    printf("[I] data load done.\n");
}

typedef double dtype;
int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    // load_data_fft<dtype>(h_x, h_gt, M, N);
    h_y = new dtype[M * N];
    // h_y = new dtype[M * N * 2];

    double total_time = 0;
    for (int i = 0; i < NUM_RUNS; ++i)
    {
        dct_2d_fft<dtype>(h_x, h_y, M, N);
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        // int flag = validate_fft<dtype>(h_y, h_gt, M ,N);
        if (!flag)
        {
            printf("[I] Error! Results are incorrect.\n", flag);
            for (int i = 0; i < 4; ++i)
            {
                printf("index: %d, result: %f, GT: %f, scale: %f\n", i, h_y[i], h_gt[i], h_y[i] / h_gt[i]);
            }
        }
        printf("[D] dct 2D takes %g ms\n", (timer_stop - timer_start) * get_timer_period());
        total_time += i > 0 ? (timer_stop - timer_start) * get_timer_period() : 0;
    }

    printf("[D] dct 2D (%d * %d) takes average %g ms\n", M, N, total_time / (NUM_RUNS - 1));

    delete[] h_x;
    delete[] h_y;
    delete[] h_gt;

    return 0;
}
