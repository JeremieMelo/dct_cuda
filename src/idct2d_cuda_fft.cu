#include <hip/hip_runtime.h>

#include <cmath>
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <assert.h>
#include <hipfft/hipfft.h>

#define PI (3.141592653589793238462643383279502884197169399375105820974944592307816406286208998628034825342117067982148086513282306647093844609550582231725359408128481)
#define TPB (16)
#define NUM_RUNS (101)

#if 0
typedef float dtype;
typedef cufftReal dtypeReal;
typedef cufftComplex dtypeComplex;
#define epsilon (5e-1) //relative error
#else
typedef double dtype;
typedef hipfftDoubleReal dtypeReal;
typedef hipfftDoubleComplex dtypeComplex;
#define epsilon (1e-2) //relative error
#endif

#define checkCUDA(status)                       \
    {                                           \
        if (status != hipSuccess)              \
        {                                       \
            printf("CUDA Runtime Error: %s\n",  \
                   hipGetErrorString(status)); \
            assert(status == hipSuccess);      \
        }                                       \
    }

typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void)
{
    using namespace std::chrono;
    return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void)
{
    using namespace std::chrono;
    return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}

hr_clock_rep timer_start, timer_stop;

/// Return true if a number is power of 2
template <typename T = unsigned>
inline bool isPowerOf2(T val)
{
    return val && (val & (val - 1)) == 0;
}

inline __device__ int INDEX(const int hid, const int wid, const int N)
{
    return (hid * N + wid);
}

template <typename T>
__global__ void idct2d_postprocess_backup(const T *x, T *y, const int M, const int N, const int halfN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int index;
        int cond = (((hid & 1) == 0) << 1) | ((wid & 1) == 0);
        switch (cond)
        {
        case 0:
            index = INDEX(2 * M - (hid + 1), N - (wid + 1) / 2, halfN);
            break;
        case 1:
            index = INDEX(2 * M - (hid + 1), wid / 2, halfN);
            break;
        case 2:
            index = INDEX(hid, N - (wid + 1) / 2, halfN);
            break;
        case 3:
            index = INDEX(hid, wid / 2, halfN);
            break;
        default:
            break;
        }
        y[INDEX(hid, wid, N)] = x[index] / 4;
    }
}

template <typename T>
__global__ void idct2d_postprocess(const T *x, T *y, const int M, const int N, const int halfN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond)
        {
        case 0:
            index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
            break;
        case 1:
            index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
            break;
        case 2:
            index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
            break;
        case 3:
            index = INDEX(hid << 1, wid << 1, N);
            break;
        default:
            assert(0);
            break;
        }
        y[index] = x[INDEX(hid, wid, N)] / 4;
    }
}

inline __device__ hipfftDoubleComplex complexMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

inline __device__ hipfftComplex complexMul(const hipfftComplex &x, const hipfftComplex &y)
{
    hipfftComplex res;
    res.x = x.x * y.x - x.y * y.y;
    res.y = x.x * y.y + x.y * y.x;
    return res;
}

inline __device__ hipfftDoubleReal RealPartOfMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    return x.x * y.x - x.y * y.y;
}

inline __device__ hipfftReal RealPartOfMul(const hipfftComplex &x, const hipfftComplex &y)
{
    return x.x * y.x - x.y * y.y;
}

inline __device__ hipfftDoubleReal ImaginaryPartOfMul(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    return x.x * y.y + x.y * y.x;
}

inline __device__ hipfftReal ImaginaryPartOfMul(const hipfftComplex &x, const hipfftComplex &y)
{
    return x.x * y.y + x.y * y.x;
}

inline __device__ hipfftDoubleComplex complexAdd(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

inline __device__ hipfftComplex complexAdd(const hipfftComplex &x, const hipfftComplex &y)
{
    hipfftComplex res;
    res.x = x.x + y.x;
    res.y = x.y + y.y;
    return res;
}

inline __device__ hipfftDoubleComplex complexSubtract(const hipfftDoubleComplex &x, const hipfftDoubleComplex &y)
{
    hipfftDoubleComplex res;
    res.x = x.x - y.x;
    res.y = x.y - y.y;
    return res;
}

inline __device__ hipfftComplex complexSubtract(const hipfftComplex &x, const hipfftComplex &y)
{
    hipfftComplex res;
    res.x = x.x - y.x;
    res.y = x.y - y.y;
    return res;
}

inline __device__ hipfftDoubleComplex complexConj(const hipfftDoubleComplex &x)
{
    hipfftDoubleComplex res;
    res.x = x.x;
    res.y = -1 * x.y;
    return res;
}

inline __device__ hipfftComplex complexConj(const hipfftComplex &x)
{
    hipfftComplex res;
    res.x = x.x;
    res.y = -1 * x.y;
    return res;
}

__global__ void precomputeExpk(hipfftDoubleComplex *expkM, hipfftDoubleComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftDoubleComplex W_h_4M = make_double2(cos(PI * hid / (2 * M)), -1 * sin(PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid <= N / 2)
    {
        int wid = tid;
        hipfftDoubleComplex W_w_4N = make_double2(cos(PI * wid / (2 * N)), -1 * sin(PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

__global__ void precomputeExpk(hipfftComplex *expkM, hipfftComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftComplex W_h_4M = make_float2(__cosf((float)PI * hid / (2 * M)), -1 * __sinf((float)PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid <= N / 2)
    {
        int wid = tid;
        hipfftComplex W_w_4N = make_float2(__cosf((float)PI * wid / (2 * N)), -1 * __sinf((float)PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

template <typename T, typename TComplex>
__global__ __launch_bounds__(TPB *TPB, 10) void idct2d_preprocess(const T *input, TComplex *output, const int M, const int N,
                                                                  const int halfM, const int halfN,
                                                                  const TComplex *__restrict__ expkM, const TComplex *__restrict__ expkN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN)
    {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond)
        {
        case 0:
        {
            T tmp1;
            TComplex tmp_up;

            output[0].x = input[0];
            output[0].y = 0;

            tmp1 = input[halfN];
            tmp_up.x = tmp1;
            tmp_up.y = tmp1;
            output[halfN] = complexConj(complexMul(expkN[halfN], tmp_up));

            tmp1 = input[INDEX(halfM, 0, N)];
            tmp_up.x = tmp1;
            tmp_up.y = tmp1;
            output[INDEX(halfM, 0, halfN + 1)] = complexConj(complexMul(expkM[halfM], tmp_up));

            tmp1 = input[INDEX(halfM, halfN, N)];
            tmp_up.x = 0;
            tmp_up.y = 2 * tmp1;
            output[INDEX(halfM, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[halfM], expkN[halfN]), tmp_up));
            break;
        }

        case 1:
        {
            TComplex tmp_up;
            tmp_up.x = input[wid];
            tmp_up.y = input[N - wid];
            output[wid] = complexConj(complexMul(expkN[wid], tmp_up));

            T tmp1 = input[INDEX(halfM, wid, N)];
            T tmp2 = input[INDEX(halfM, N - wid, N)];
            tmp_up.x = tmp1 - tmp2;
            tmp_up.y = tmp1 + tmp2;
            output[INDEX(halfM, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[halfM], expkN[wid]), tmp_up));
            break;
        }

        case 2:
        {
            T tmp1, tmp3;
            TComplex tmp_up, tmp_down;

            tmp1 = input[INDEX(hid, 0, N)];
            tmp3 = input[INDEX(M - hid, 0, N)];
            tmp_up.x = tmp1;
            tmp_up.y = tmp3;
            tmp_down.x = tmp3;
            tmp_down.y = tmp1;

            output[INDEX(hid, 0, halfN + 1)] = complexConj(complexMul(expkM[hid], tmp_up));
            output[INDEX(M - hid, 0, halfN + 1)] = complexConj(complexMul(expkM[M - hid], tmp_down));

            tmp1 = input[INDEX(hid, halfN, N)];
            tmp3 = input[INDEX(M - hid, halfN, N)];
            tmp_up.x = tmp1 - tmp3;
            tmp_up.y = tmp3 + tmp1;
            tmp_down.x = tmp3 - tmp1;
            tmp_down.y = tmp1 + tmp3;

            output[INDEX(hid, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[hid], expkN[halfN]), tmp_up));
            output[INDEX(M - hid, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[M - hid], expkN[halfN]), tmp_down));
            break;
        }

        case 3:
        {
            T tmp1 = input[INDEX(hid, wid, N)];
            T tmp2 = input[INDEX(hid, N - wid, N)];
            T tmp3 = input[INDEX(M - hid, wid, N)];
            T tmp4 = input[INDEX(M - hid, N - wid, N)];
            TComplex tmp_up, tmp_down;
            tmp_up.x = tmp1 - tmp4;
            tmp_up.y = tmp3 + tmp2;
            tmp_down.x = tmp3 - tmp2;
            tmp_down.y = tmp1 + tmp4;

            output[INDEX(hid, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[hid], expkN[wid]), tmp_up));
            output[INDEX(M - hid, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[M - hid], expkN[wid]), tmp_down));
            break;
        }

        default:
            assert(0);
            break;
        }
    }
}

template <typename T>
void makeCufftPlan(const int M, const int N, hipfftHandle *plan) {}

template <>
void makeCufftPlan<hipfftComplex>(const int M, const int N, hipfftHandle *plan)
{
    hipfftPlan2d(plan, M, N, HIPFFT_C2R);
}

template <>
void makeCufftPlan<hipfftDoubleComplex>(const int M, const int N, hipfftHandle *plan)
{
    hipfftPlan2d(plan, M, N, HIPFFT_Z2D);
}

void ifft2D(hipfftDoubleComplex *d_x, hipfftDoubleReal *d_y, const int M, const int N, hipfftHandle &plan)
{
    hipfftExecZ2D(plan, d_x, d_y);
    hipDeviceSynchronize();
}

void ifft2D(hipfftComplex *d_x, hipfftReal *d_y, const int M, const int N, hipfftHandle &plan)
{
    hipfftExecC2R(plan, d_x, d_y);
    hipDeviceSynchronize();
}

template <typename T, typename TReal = hipfftDoubleReal, typename TComplex = hipfftDoubleComplex>
void dct_2d_fft(const T *h_x, T *h_y, const int M, const int N)
{
    T *d_x;
    T *d_y;
    TComplex *scratch;
    TComplex *expkM, *expkN;

    if (!isPowerOf2<int>(N) || !isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = M * N * sizeof(T);
    checkCUDA(hipMalloc((void **)&d_x, size));
    checkCUDA(hipMalloc((void **)&d_y, size));
    checkCUDA(hipMalloc((void **)&expkM, M * sizeof(TComplex)));
    checkCUDA(hipMalloc((void **)&expkN, (N / 2 + 1) * sizeof(TComplex)));
    checkCUDA(hipMalloc((void **)&scratch, M * (N / 2 + 1) * sizeof(TComplex)));
    checkCUDA(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));

    hipfftHandle plan;
    makeCufftPlan<TComplex>(M, N, &plan);

    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 gridSize2((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    precomputeExpk<<<(std::max(M, N) + 1023) / 1024, 1024>>>(expkM, expkN, M, N);
    hipDeviceSynchronize();

    timer_start = get_globaltime();
    idct2d_preprocess<T, TComplex><<<gridSize2, blockSize>>>(d_x, scratch, M, N, M / 2, N / 2, expkM, expkN);
    hipDeviceSynchronize();

    ifft2D(scratch, d_x, M, N, plan);

    idct2d_postprocess<T><<<gridSize, blockSize>>>(d_x, d_y, M, N, N / 2);
    hipDeviceSynchronize();
    timer_stop = get_globaltime();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(scratch);
    hipFree(expkM);
    hipFree(expkN);
    hipfftDestroy(plan);
}

template <typename T>
int validate2D(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag;
            if (std::abs(result_gt[i * N + j]) < 1e-6)
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j])) < epsilon / 100.;
            }
            else
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j]) / std::abs(result_gt[i * N + j])) < epsilon;
            }
            if (flag == 0)
            {
                printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i * N + j], i, j, result_gt[i * N + j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
T **allocateMatrix(int M, int N)
{
    T **data;
    data = new T *[M];
    for (int i = 0; i < M; i++)
    {
        data[i] = new T[N];
    }
    return data;
}

template <typename T>
void destroyMatrix(T **&data, int M)
{
    for (int i = 0; i < M; i++)
    {
        delete[] data[i];
    }
    delete[] data;
}

template <typename T>
void load_data(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("result_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("test_2d.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N];
    while (input_file2 >> val)
    {
        result[i] = val;
        i++;
    }
    printf("[I] data load done.\n");
}

int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    h_y = new dtype[M * N];

    double total_time = 0;
    for (int i = 0; i < NUM_RUNS; ++i)
    {
        dct_2d_fft<dtype, dtypeReal, dtypeComplex>(h_x, h_y, M, N);
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        if (!flag)
        {
            printf("[I] Error! Results are incorrect.\n", flag);
            for (int i = 0; i < 64; ++i)
            {
                printf("index: %d, result: %f, GT: %f, scale: %f\n", i, h_y[i], h_gt[i], h_y[i] / h_gt[i]);
            }
        }
        printf("[D] idct 2D takes %g ms\n", (timer_stop - timer_start) * get_timer_period());
        total_time += i > 0 ? (timer_stop - timer_start) * get_timer_period() : 0;
    }

    printf("[D] idct 2D (%d * %d) takes average %g ms\n", M, N, total_time / (NUM_RUNS - 1));

    delete[] h_x;
    delete[] h_y;
    delete[] h_gt;

    return 0;
}
