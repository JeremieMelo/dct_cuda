#include "hip/hip_runtime.h"
// idct(idxst(x)) is similar to the idct2d(x),
// except tiny modification on preprocessing and postprocessing
#include "global.cuh"

#define TPB (16)

// Adpated from idct2d_postprocess() with changes on sign and scale
// if (wid % 2 == 1)
//     new_output[hid][wid] = -output[hid][wid];
// else
//     new_output[hid][wid] = output[hid][wid];
template <typename T>
__global__ void idct_idxst_postprocess_backup(const T *x, T *y, const int M, const int N, const int halfN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int index;
        int cond = (((hid & 1) == 0) << 1) | ((wid & 1) == 0);
        switch (cond)
        {
        case 0:
            index = INDEX(2 * M - (hid + 1), N - (wid + 1) / 2, halfN);
            y[INDEX(hid, wid, N)] = -x[index];
            break;
        case 1:
            index = INDEX(2 * M - (hid + 1), wid / 2, halfN);
            y[INDEX(hid, wid, N)] = x[index];
            break;
        case 2:
            index = INDEX(hid, N - (wid + 1) / 2, halfN);
            y[INDEX(hid, wid, N)] = -x[index];
            break;
        case 3:
            index = INDEX(hid, wid / 2, halfN);
            y[INDEX(hid, wid, N)] = x[index];
            break;
        default:
            assert(0);
            break;
        }
    }
}

// Adpated from idct2d_postprocess() with changes on sign and scale
// if (wid % 2 == 1)
//     new_output[hid][wid] = -output[hid][wid];
// else
//     new_output[hid][wid] = output[hid][wid];
template <typename T>
__global__ void idct_idxst_postprocess(const T *x, T *y, const int M, const int N, const int halfN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < M && wid < N)
    {
        int cond = ((hid < M / 2) << 1) | (wid < N / 2);
        int index;
        switch (cond)
        {
        case 0:
            index = INDEX(((M - hid) << 1) - 1, ((N - wid) << 1) - 1, N);
            y[index] = -x[INDEX(hid, wid, N)];
            break;
        case 1:
            index = INDEX(((M - hid) << 1) - 1, wid << 1, N);
            y[index] = x[INDEX(hid, wid, N)];
            break;
        case 2:
            index = INDEX(hid << 1, ((N - wid) << 1) - 1, N);
            y[index] = -x[INDEX(hid, wid, N)];
            break;
        case 3:
            index = INDEX(hid << 1, wid << 1, N);
            y[index] = x[INDEX(hid, wid, N)];
            break;
        default:
            assert(0);
            break;
        }
    }
}

__global__ void precomputeExpk(hipfftDoubleComplex *expkM, hipfftDoubleComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftDoubleComplex W_h_4M = make_double2(cos(PI * hid / (2 * M)), -1 * sin(PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid <= N / 2)
    {
        int wid = tid;
        hipfftDoubleComplex W_w_4N = make_double2(cos(PI * wid / (2 * N)), -1 * sin(PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

__global__ void precomputeExpk(hipfftComplex *expkM, hipfftComplex *expkN, const int M, const int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M)
    {
        int hid = tid;
        hipfftComplex W_h_4M = make_float2(__cosf((float)PI * hid / (2 * M)), -1 * __sinf((float)PI * hid / (M * 2)));
        expkM[hid] = W_h_4M;
    }
    if (tid <= N / 2)
    {
        int wid = tid;
        hipfftComplex W_w_4N = make_float2(__cosf((float)PI * wid / (2 * N)), -1 * __sinf((float)PI * wid / (N * 2)));
        expkN[wid] = W_w_4N;
    }
}

// Adpated from idct2d_preprocess(). The only change is the reordered input
// if (wid != 0)
//     new_input[hid][wid] = input[hid][N - wid];
// else
//     new_input[hid][0] = 0
template <typename T, typename TComplex>
__global__ __launch_bounds__(TPB *TPB, 10) void idct_idxst_preprocess(const T *input, TComplex *output, const int M, const int N,
                                                                      const int halfM, const int halfN,
                                                                      const TComplex *__restrict__ expkM, const TComplex *__restrict__ expkN)
{
    const int wid = blockDim.x * blockIdx.x + threadIdx.x;
    const int hid = blockDim.y * blockIdx.y + threadIdx.y;
    if (hid < halfM && wid < halfN)
    {
        int cond = ((hid != 0) << 1) | (wid != 0);
        switch (cond)
        {
        case 0:
        {
            T tmp1;
            TComplex tmp_up;

            output[0].x = 0;
            output[0].y = 0;

            tmp1 = input[halfN];
            tmp_up.x = tmp1;
            tmp_up.y = tmp1;
            output[halfN] = complexConj(complexMul(expkN[halfN], tmp_up));

            output[INDEX(halfM, 0, halfN + 1)].x = 0;
            output[INDEX(halfM, 0, halfN + 1)].y = 0;

            tmp1 = input[INDEX(halfM, halfN, N)];
            tmp_up.x = 0;
            tmp_up.y = 2 * tmp1;
            output[INDEX(halfM, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[halfM], expkN[halfN]), tmp_up));
            break;
        }

        case 1:
        {
            TComplex tmp_up;
            tmp_up.x = input[N - wid];
            tmp_up.y = input[wid];
            output[wid] = complexConj(complexMul(expkN[wid], tmp_up));

            T tmp1 = input[INDEX(halfM, N - wid, N)];
            T tmp2 = input[INDEX(halfM, wid, N)];
            tmp_up.x = tmp1 - tmp2;
            tmp_up.y = tmp1 + tmp2;
            output[INDEX(halfM, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[halfM], expkN[wid]), tmp_up));
            break;
        }

        case 2:
        {
            T tmp1, tmp3;
            TComplex tmp_up, tmp_down;

            output[INDEX(hid, 0, halfN + 1)].x = 0;
            output[INDEX(hid, 0, halfN + 1)].y = 0;
            output[INDEX(M - hid, 0, halfN + 1)].x = 0;
            output[INDEX(M - hid, 0, halfN + 1)].y = 0;

            tmp1 = input[INDEX(hid, halfN, N)];
            tmp3 = input[INDEX(M - hid, halfN, N)];
            tmp_up.x = tmp1 - tmp3;
            tmp_up.y = tmp3 + tmp1;
            tmp_down.x = tmp3 - tmp1;
            tmp_down.y = tmp1 + tmp3;

            output[INDEX(hid, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[hid], expkN[halfN]), tmp_up));
            output[INDEX(M - hid, halfN, halfN + 1)] = complexConj(complexMul(complexMul(expkM[M - hid], expkN[halfN]), tmp_down));
            break;
        }

        case 3:
        {
            T tmp1 = input[INDEX(hid, N - wid, N)];
            T tmp2 = input[INDEX(hid, wid, N)];
            T tmp3 = input[INDEX(M - hid, N - wid, N)];
            T tmp4 = input[INDEX(M - hid, wid, N)];
            TComplex tmp_up, tmp_down;
            tmp_up.x = tmp1 - tmp4;
            tmp_up.y = tmp3 + tmp2;
            tmp_down.x = tmp3 - tmp2;
            tmp_down.y = tmp1 + tmp4;

            output[INDEX(hid, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[hid], expkN[wid]), tmp_up));
            output[INDEX(M - hid, wid, halfN + 1)] = complexConj(complexMul(complexMul(expkM[M - hid], expkN[wid]), tmp_down));
            break;
        }

        default:
            assert(0);
            break;
        }
    }
}

template <typename T>
void makeCufftPlan(const int M, const int N, hipfftHandle *plan) {}

template <>
void makeCufftPlan<hipfftComplex>(const int M, const int N, hipfftHandle *plan)
{
    hipfftPlan2d(plan, M, N, HIPFFT_C2R);
}

template <>
void makeCufftPlan<hipfftDoubleComplex>(const int M, const int N, hipfftHandle *plan)
{
    hipfftPlan2d(plan, M, N, HIPFFT_Z2D);
}

void ifft2D(hipfftDoubleComplex *d_x, hipfftDoubleReal *d_y, hipfftHandle &plan)
{
    hipfftExecZ2D(plan, d_x, d_y);
    hipDeviceSynchronize();
}

void ifft2D(hipfftComplex *d_x, hipfftReal *d_y, hipfftHandle &plan)
{
    hipfftExecC2R(plan, d_x, d_y);
    hipDeviceSynchronize();
}

CpuTimer Timer;
template <typename T, typename TReal = hipfftDoubleReal, typename TComplex = hipfftDoubleComplex>
void idct_idxst(const T *h_x, T *h_y, const int M, const int N)
{
    T *d_x;
    T *d_y;
    T *ifft_result;
    TComplex *scratch;
    TComplex *expkM, *expkN;

    if (!isPowerOf2<int>(N) || !isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = M * N * sizeof(T);
    cudaSafeCall(hipMalloc((void **)&d_x, size));
    cudaSafeCall(hipMalloc((void **)&d_y, size));
    cudaSafeCall(hipMalloc((void **)&ifft_result, size));
    cudaSafeCall(hipMalloc((void **)&expkM, M * sizeof(TComplex)));
    cudaSafeCall(hipMalloc((void **)&expkN, (N / 2 + 1) * sizeof(TComplex)));
    cudaSafeCall(hipMalloc((void **)&scratch, M * (N / 2 + 1) * sizeof(TComplex)));
    cudaSafeCall(hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice));

    hipfftHandle plan;
    makeCufftPlan<TComplex>(M, N, &plan);

    dim3 gridSize((N + TPB - 1) / TPB, (M + TPB - 1) / TPB, 1);
    dim3 gridSize2((N / 2 + TPB - 1) / TPB, (M / 2 + TPB - 1) / TPB, 1);
    dim3 blockSize(TPB, TPB, 1);
    precomputeExpk<<<(std::max(M, N) + 1023) / 1024, 1024>>>(expkM, expkN, M, N);
    hipDeviceSynchronize();

    Timer.Start();
    idct_idxst_preprocess<T, TComplex><<<gridSize2, blockSize>>>(d_x, scratch, M, N, M / 2, N / 2, expkM, expkN);
    hipDeviceSynchronize();

    ifft2D(scratch, ifft_result, plan);

    idct_idxst_postprocess<T><<<gridSize, blockSize>>>(ifft_result, d_y, M, N, N / 2);
    hipDeviceSynchronize();
    Timer.Stop();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(ifft_result);
    hipFree(scratch);
    hipFree(expkM);
    hipFree(expkN);
    hipfftDestroy(plan);
}

template <typename T>
int validate2D(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag;
            if (std::abs(result_gt[i * N + j]) < 1e-6)
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j])) < epsilon / 100.;
            }
            else
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j]) / std::abs(result_gt[i * N + j])) < epsilon;
            }
            if (flag == 0)
            {
                printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i * N + j], i, j, result_gt[i * N + j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
void load_data(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("idct_idxst.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N];
    while (input_file2 >> val)
    {
        result[i] = val * 2; // scale factor
        i++;
    }
    printf("[I] data load done.\n");
}

int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    h_y = new dtype[M * N];

    double total_time = 0;
    for (int i = 0; i < NUM_RUNS; ++i)
    {
        idct_idxst<dtype, dtypeReal, dtypeComplex>(h_x, h_y, M, N);
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        if (!flag)
        {
            printf("[I] Error! Results are incorrect.\n", flag);
            for (int i = 0; i < 64; ++i)
            {
                printf("index: %d, result: %f, GT: %f, scale: %f\n", i, h_y[i], h_gt[i], h_y[i] / h_gt[i]);
            }
        }
        printf("[D] idct_idxst takes %g ms\n", Timer.ElapsedMillis());
        total_time += i > 0 ? Timer.ElapsedMillis() : 0;
    }

    printf("[D] idct_idxst (%d * %d) takes average %g ms\n", M, N, total_time / (NUM_RUNS - 1));

    delete[] h_x;
    delete[] h_y;
    delete[] h_gt;

    return 0;
}
