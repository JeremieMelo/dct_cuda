#include "hip/hip_runtime.h"
#include "global.cuh"

#define TPB (1024)

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename T>
__global__ void precompute_dct_cos_kernel_backup(T *d_cos, T *scratch, int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N - 1)
    {
        int sum = N / 2;
        int halfLen = N / 2;
        while (tid >= sum)
        {
            halfLen = halfLen / 2;
            sum += halfLen;
        }
        T phase = (0.5 + tid - (sum - halfLen)) * PI / (halfLen << 1);
        d_cos[tid] = 0.5 / cos(phase);
    }
    else if (tid == N - 1)
    {
        d_cos[tid] = 0;
    }
}

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename T>
__global__ void precompute_dct_cos_kernel(T *d_cos, int N, int log_N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int total_height = log_N;
    if (tid < N - 1)
    {
        int k = N - tid - 1;
        // int total_height = LogBase2(N);
        int height = LogBase2(k);
        // int len = N / (1 << (total_height - height - 1));
        int len = 1 << (height + 1);
        int i = len - k - 1;

        T phase = (0.5 + i) * PI / len;
        d_cos[tid] = 0.5 / cos(phase);
    }
    else if (tid == N - 1)
    {
        d_cos[tid] = 0;
    }
}

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename T>
void precompute_dct_cos(T *cos, int N)
{
    // The input length must be power of 2
    if (!isPowerOf2<int>(N))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    // create the array on host
    T *cos_host = new T[N];

    int offset = 0;
    int halfLen = N / 2;
    while (halfLen)
    {
        T phaseStep = PI / (halfLen << 1);
        // T phase_start = 0.5 * phaseStep;
        // #pragma omp parallel for
        for (int i = 0; i < halfLen; ++i)
        {
            T phase = (0.5 + i) * phaseStep;
            cos_host[offset + i] = 0.5 / std::cos(phase);
        }
        offset += halfLen;
        halfLen >>= 1;
    }
    // printf("last cos: %f\n", cos_host[N-1]);
    // while (halfLen)
    // {
    //     T phaseStep = 0.5 * PI / halfLen;
    //     T phase = 0.5 * phaseStep;
    //     for (int i = 0; i < halfLen; ++i)
    //     {
    //         cos_host[offset + i] = 0.5 / std::cos(phase);
    //         phase += phaseStep;
    //     }
    //     offset += halfLen;
    //     halfLen /= 2;
    // }

    // copy to GPU
    hipMemcpy(cos, cos_host, N * sizeof(T), hipMemcpyHostToDevice);

    delete[] cos_host;
}

template <typename T, typename TIndex>
__global__ void computeDctForward_1(const T *__restrict__ curr_ptr, T *next_ptr, const T *cos, TIndex N, TIndex len, TIndex halfLen, TIndex cosOffset)
{
    TIndex halfN = (N >> 1);
    TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < halfN)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2 + blockIdx.y * N;
        T *next = next_ptr + offset + i;
        const T *__restrict__ curr = curr_ptr + offset;

        next[0] = curr[i] + curr[len - i - 1];
        next[halfLen] = (curr[i] - curr[len - i - 1]) * cos[cosOffset + i];
    }
}

template <typename T, typename TIndex>
__global__ void computeDctBackward_1(const T *__restrict__ curr_ptr, T *next_ptr, TIndex N, TIndex len, TIndex halfLen)
{
    TIndex halfN = (N >> 1);
    TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < halfN)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2 + blockIdx.y * N;
        T *next = next_ptr + offset + i * 2;
        const T *__restrict__ curr = curr_ptr + offset;

        next[0] = curr[i];
        next[1] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
    }
}

template <typename T, typename TIndex>
__global__ void computeDctBackward_lasttime_1(const T *__restrict__ curr_ptr, T *next_ptr, TIndex M, TIndex N, TIndex len, TIndex halfLen)
{
    TIndex halfN = (N >> 1);
    TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < halfN)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;
        T *next = next_ptr + blockIdx.y + (offset + i * 2) * M;
        const T *__restrict__ curr = curr_ptr + offset + blockIdx.y * N;

        next[0] = curr[i];
        next[M] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
    }
}

#define ROW2COL(IDX, COL, N) ((IDX) * (N) + (COL))

template <typename T, typename TIndex>
__global__ void computeDctForward_2(const T *__restrict__ curr, T *next, const T *cos, TIndex M, TIndex N, TIndex len, TIndex halfLen, TIndex cosOffset)
{
    TIndex halfM = (M >> 1);
    TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < halfM)
    {
        TIndex col = blockIdx.y;
        TIndex rest = thread_id & (halfM - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[ROW2COL(offset + i, col, N)] = curr[ROW2COL(offset + i, col, N)] + curr[ROW2COL(offset + len - i - 1, col, N)];
        next[ROW2COL(offset + i + halfLen, col, N)] = (curr[ROW2COL(offset + i, col, N)] - curr[ROW2COL(offset + len - i - 1, col, N)]) * cos[cosOffset + i];
    }
}

template <typename T, typename TIndex>
__global__ void computeDctBackward_2(const T *__restrict__ curr, T *next, TIndex M, TIndex N, TIndex len, TIndex halfLen)
{
    TIndex halfM = (M >> 1);
    TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    if (thread_id < halfM)
    {
        TIndex col = blockIdx.y;
        TIndex rest = thread_id & (halfM - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[ROW2COL(offset + i * 2, col, N)] = curr[ROW2COL(offset + i, col, N)];
        next[ROW2COL(offset + i * 2 + 1, col, N)] = (i + 1 == halfLen) ? curr[ROW2COL(offset + len - 1, col, N)] : curr[ROW2COL(offset + halfLen + i, col, N)] + curr[ROW2COL(offset + halfLen + i + 1, col, N)];
    }
}

template <typename T>
__global__ void normalize(T *x, const T *__restrict__ y, const int M, const int N)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M * N)
    {
        x[tid] = y[tid] / (M * N) * 4;
    }
}

template <typename T>
__global__ void normalize4(T *x, const T *__restrict__ y, const int size, T factor)
{
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    double4 *tmp_x = (double4 *)x;
    const double4 *__restrict__ tmp_y = (const double4 *)y;
    tmp_x[tid] = make_double4(tmp_y[tid].x * factor, tmp_y[tid].y * factor, tmp_y[tid].z * factor, tmp_y[tid].w * factor);
}

/// The implementation of fast Discrete Cosine Transform (DCT) algorithm and its inverse (IDCT) are Lee's algorithms
/// Algorithm reference: A New Algorithm to Compute the Discrete Cosine Transform, by Byeong Gi Lee, 1984
///
/// Lee's algorithm has a recursive structure in nature.
/// Here is a sample recursive implementation: https://www.nayuki.io/page/fast-discrete-cosine-transform-algorithms
///
/// My implementation here is iterative, which is more efficient than the recursive version.
/// Here is a sample iterative implementation: https://www.codeproject.com/Articles/151043/Iterative-Fast-1D-Forvard-DCT

/// Compute y[k] = sum_n=0..N-1 (x[n] * cos((n + 0.5) * k * PI / N)), for k = 0..N-1
///
/// @param  vec   length M * N sequence to be transformed in last dimension
/// @param  out   length M * N helping buffer, which is also the output
/// @param  buf   length M * N helping buffer
/// @param  cos   length N - 1, stores cosine values precomputed by function 'precompute_dct_cos'
/// @param  M     length of dimension 0 of vec
/// @param  N     length of dimension 1 of vec, must be power of 2
template <typename T>
void dct_ref_1(const T *vec, T *out, T *buf, const T *cos, int M, int N)
{
    dim3 gridSize((N / 2 + TPB - 1) / TPB, M, 1);
    dim3 blockSize(TPB, 1, 1);

    // Pointers point to the beginning indices of two adjacent iterations
    T *curr = buf;
    T *next = out;

    // 'temp' used to store date of two adjacent iterations
    // Copy 'vec' to the first N element in 'temp'
    hipMemcpy(curr, vec, M * N * sizeof(T), hipMemcpyDeviceToDevice);

    // Current bufferfly length and half length
    int len = N;
    int halfLen = len / 2;

    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;
    while (halfLen)
    {
        computeDctForward_1<<<gridSize, blockSize>>>(curr, next, cos, N, len, halfLen, cosOffset);
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        swap(curr, next);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (halfLen < N)
    {
        computeDctBackward_1<<<gridSize, blockSize>>>(curr, next, N, len, halfLen);
        halfLen = len;
        len *= 2;
        swap(curr, next);
    }

    // Populate the final results into 'out'
    if (curr != out)
    {
        swap(out, buf);
    }
}

template <typename T>
void dct_ref_2(const T *vec, T *out, T *buf, const T *cos, int M, int N)
{
    dim3 gridSize((M / 2 + TPB - 1) / TPB, N, 1);
    dim3 blockSize(TPB, 1, 1);
    // int block_count = (N + TPB - 1) / TPB;
    // int thread_count = TPB;

    // Pointers point to the beginning indices of two adjacent iterations
    T *curr = buf;
    T *next = out;

    // 'temp' used to store date of two adjacent iterations
    // Copy 'vec' to the first N element in 'temp'
    hipMemcpy(curr, vec, M * N * sizeof(T), hipMemcpyDeviceToDevice);

    // Current bufferfly length and half length
    int len = M;
    int halfLen = len / 2;

    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;
    while (halfLen)
    {
        computeDctForward_2<<<gridSize, blockSize>>>(curr, next, cos, M, N, len, halfLen, cosOffset);
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        swap(curr, next);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (halfLen < M)
    {
        computeDctBackward_2<<<gridSize, blockSize>>>(curr, next, M, N, len, halfLen);
        halfLen = len;
        len *= 2;
        swap(curr, next);
    }

    // Populate the final results into 'out'
    normalize<T><<<(N * M + TPB - 1) / TPB, TPB>>>(out, curr, M, N);
}

template <typename T, typename TIndex>
__global__ __launch_bounds__(1024, 10) void dct_1d_kernel(const T *__restrict__ vec, T *out, const T *cos, const int N, const T factor)
{
    extern __shared__ T sdata[];
    T *curr_ptr = sdata;
    T *next_ptr = curr_ptr + N;

    for (TIndex i = threadIdx.x; i < N; i += blockDim.x)
    {
        curr_ptr[i] = vec[blockIdx.y * N + i];
    }
    __syncthreads();

    // Current bufferfly length and half length
    int len = N;
    int halfLen = len / 2;
    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;

    const TIndex halfN = halfLen;
    while (halfLen)
    {
        #pragma unroll 2
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i;
            T *curr = curr_ptr + offset;

            next[0] = curr[i] + curr[len - i - 1];
            next[halfLen] = (curr[i] - curr[len - i - 1]) * cos[cosOffset + i];
        }
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (len < N)
    {
        #pragma unroll 2
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i * 2;
            T *curr = curr_ptr + offset;

            T tmp1 = curr[i];
            T tmp2 = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];

            *(double2 *)next = make_double2(tmp1, tmp2);
            // next[0] = curr[i];
            // next[1] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
        }
        halfLen = len;
        len *= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }
    #pragma unroll 2
    for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;
        T *next = out + offset + i * 2;
        T *curr = curr_ptr + offset;

        T tmp1 = curr[i] * factor;
        T tmp2 = ((i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1]) * factor;
        *(double2 *)next = make_double2(tmp1, tmp2);
        // next[0] = curr[i];
        // next[1] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
    }
    __syncthreads();
}

template <typename T, typename TIndex>
__global__ __launch_bounds__(1024, 10) void dct_transpose_kernel(const T *__restrict__ vec, T *out, const T *cos, const int M, const int N)
{
    extern __shared__ T sdata[];
    T *curr_ptr = sdata;
    T *next_ptr = curr_ptr + N;

    for (TIndex i = threadIdx.x; i < N; i += blockDim.x)
    {
        curr_ptr[i] = vec[blockIdx.y * N + i];
    }
    __syncthreads();

    // Current bufferfly length and half length
    int len = N;
    int halfLen = len / 2;
    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;

    const TIndex halfN = halfLen;
    while (halfLen)
    {
        #pragma unroll 2
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i;
            T *curr = curr_ptr + offset;

            next[0] = curr[i] + curr[len - i - 1];
            next[halfLen] = (curr[i] - curr[len - i - 1]) * cos[cosOffset + i];
        }
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (len < N)
    {
        #pragma unroll 2
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i * 2;
            T *curr = curr_ptr + offset;

            T tmp1 = curr[i];
            T tmp2 = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];

            *(double2 *)next = make_double2(tmp1, tmp2);
            // next[0] = curr[i];
            // next[1] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
        }
        halfLen = len;
        len *= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }
    #pragma unroll 2
    for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;
        T *next = out + blockIdx.y + (offset + i * 2) * M;
        T *curr = curr_ptr + offset;

        next[0] = curr[i];
        next[M] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
    }
    __syncthreads();
}

template <typename T, typename TIndex>
__global__ void dct_transpose_normalize_kernel(const T *__restrict__ vec, T *out, const T *cos, const int M, const int N)
{
    extern __shared__ T sdata[];
    T *curr_ptr = sdata;
    T *next_ptr = curr_ptr + N;

    for (TIndex i = threadIdx.x; i < N; i += blockDim.x)
    {
        curr_ptr[i] = vec[blockIdx.y * N + i];
    }
    __syncthreads();

    // Current bufferfly length and half length
    int len = N;
    int halfLen = len / 2;
    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;

    const TIndex halfN = halfLen;
    while (halfLen)
    {
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i;
            T *curr = curr_ptr + offset;

            next[0] = curr[i] + curr[len - i - 1];
            next[halfLen] = (curr[i] - curr[len - i - 1]) * cos[cosOffset + i];
        }
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (len < N)
    {
        for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
        {
            TIndex rest = thread_id & (halfN - 1);
            TIndex i = rest & (halfLen - 1);
            TIndex offset = (thread_id - i) * 2;
            T *next = next_ptr + offset + i * 2;
            T *curr = curr_ptr + offset;

            next[0] = curr[i];
            next[1] = (i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1];
        }
        halfLen = len;
        len *= 2;
        __syncthreads();
        swap(curr_ptr, next_ptr);
    }

    for (TIndex thread_id = threadIdx.x; thread_id < halfN; thread_id += blockDim.x)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;
        T *next = out + blockIdx.y + (offset + i * 2) * M;
        T *curr = curr_ptr + offset;

        next[0] = curr[i] / (M * N) * 4;
        next[M] = ((i + 1 == halfLen) ? curr[len - 1] : curr[halfLen + i] + curr[halfLen + i + 1]) / (M * N) * 4;
    }
    __syncthreads();
}

template <typename T>
void dct_transpose(const T *vec, T *out, const T *cos, int M, int N)
{
    dim3 gridSize(1, M, 1);
    dim3 blockSize(std::min(TPB, N >> 1), 1, 1);
    size_t shared_memory_size = 2 * N * sizeof(T);
    dct_transpose_kernel<T, int><<<gridSize, blockSize, shared_memory_size>>>(vec, out, cos, M, N);
}

template <typename T>
void dct_transpose_normalize(const T *vec, T *out, const T *cos, int M, int N)
{
    dim3 gridSize(1, M, 1);
    dim3 blockSize(std::min(TPB, N >> 1), 1, 1);
    size_t shared_memory_size = 2 * N * sizeof(T);
    dct_transpose_normalize_kernel<T, int><<<gridSize, blockSize, shared_memory_size>>>(vec, out, cos, M, N);
}

CpuTimer Timer;

template <typename T>
void dct_1d_lee(const T *h_x, T *h_y, const int N)
{
    T *d_x;
    T *d_y;
    T *d_cos0;

    if (!isPowerOf2<int>(N))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = N * sizeof(T);
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&d_cos0, N * sizeof(T)); // row

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    precompute_dct_cos_kernel<T><<<(N + TPB - 1) / TPB, TPB, 0, streams[0]>>>(d_cos0, N, (int)log2(N));
    dim3 gridSize(1, 1, 1);
    dim3 blockSize(std::min(TPB, N >> 1), 1, 1);
    size_t shared_memory_size = 2 * N * sizeof(T);

    hipDeviceSynchronize();
    Timer.Start();

    dct_1d_kernel<T, int><<<gridSize, blockSize, shared_memory_size>>>(d_x, d_y, d_cos0, N, 2.0 / N);

    hipDeviceSynchronize();
    Timer.Stop();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_cos0);
}

template <typename T>
void dct_2d_lee(const T *h_x, T *h_y, const int M, const int N)
{
    T *d_x;
    T *d_y;
    T *scratch;
    T *d_cos0;
    T *d_cos1;

    if (!isPowerOf2<int>(N) || !isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0);
    }

    size_t size = M * N * sizeof(T);
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&scratch, size);
    hipMalloc((void **)&d_cos0, N * sizeof(T)); // row
    hipMalloc((void **)&d_cos1, M * sizeof(T)); // column

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);

    hipStream_t streams[2];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);

    precompute_dct_cos_kernel<T><<<(N + TPB - 1) / TPB, TPB, 0, streams[0]>>>(d_cos0, N, (int)log2(N));
    precompute_dct_cos_kernel<T><<<(M + TPB - 1) / TPB, TPB, 0, streams[1]>>>(d_cos1, M, (int)log2(M));
    hipDeviceSynchronize();

    Timer.Start();

    #if 1
    dct_transpose<T>(d_x, scratch, d_cos0, M, N);
    dct_transpose<T>(scratch, d_y, d_cos1, N, M);
    // normalize<T><<<(N * M + TPB - 1) / TPB, TPB>>>(d_y, d_y, M, N);
    normalize4<T><<<(N * M / 4 + TPB - 1) / TPB, TPB>>>(d_y, d_y, M * N / 4, 4. / (M * N));
    #elif 1
    dct_transpose<T>(d_x, scratch, d_cos0, M, N);
    dct_transpose_normalize<T>(scratch, d_y, d_cos1, N, M);
    #elif 0
    dct_ref_1<T>(d_x, d_y, scratch, d_cos0, M, N);
    transpose<T>(d_y, scratch, M, N);
    dct_ref_1<T>(scratch, d_y, scratch, d_cos1, N, M);
    transpose<T>(d_y, scratch, N, M);
    normalize<T><<<(N * M + TPB - 1) / TPB, TPB>>>(d_y, scratch, M, N);
    #elif 1
    dct_ref_1<T>(d_x, d_y, scratch, d_cos0, M, N);
    dct_ref_2<T>(d_y, d_y, scratch, d_cos1, M, N);
    #endif

    hipDeviceSynchronize();
    Timer.Stop();

    hipMemcpy(h_y, d_y, size, hipMemcpyDeviceToHost);

    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(scratch);
    hipFree(d_cos0);
    hipFree(d_cos1);
}

template <typename T>
int validate2D(T *result_cuda, T *result_gt, const int M, const int N)
{
    for (int i = 0; i < M; ++i)
    {
        for (int j = 0; j < N; ++j)
        {
            int flag;
            if (std::abs(result_gt[i * N + j]) < 1e-6)
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j])) < epsilon / 100.;
            }
            else
            {
                flag = (std::abs(result_cuda[i * N + j] - result_gt[i * N + j]) / std::abs(result_gt[i * N + j])) < epsilon;
            }
            if (flag == 0)
            {
                printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i * N + j], i, j, result_gt[i * N + j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
void load_data(T *&data, T *&result, int &M, int &N)
{
    std::ifstream input_file("test_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T[M * N];
    while (input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T[M * N];
    while (input_file2 >> val)
    {
        result[i] = val;
        i++;
    }
    printf("[I] data load done.\n");
}

int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    h_y = new dtype[M * N];

    double total_time = 0;
    for (int i = 0; i < NUM_RUNS; ++i)
    {
        if (M == 1)
        {
            dct_1d_lee<dtype>(h_x, h_y, N);
        }
        else
        {
            dct_2d_lee<dtype>(h_x, h_y, M, N);
        }
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        if (!flag)
        {
            printf("[I] Error! Results are incorrect.\n", flag);
            for (int i = 0; i < 5; ++i)
            {
                printf("index: %d, result: %f, GT: %f\n", i, h_y[i], h_gt[i]);
            }
        }
        printf("[D] dct 2D takes %g ms\n", Timer.ElapsedMillis());
        total_time += i > 0 ? Timer.ElapsedMillis() : 0;
    }

    printf("[D] dct 2D (%d * %d) takes average %g ms\n", M, N, total_time / NUM_RUNS);

    delete[] h_x;
    delete[] h_y;
    delete[] h_gt;

    return 0;
}
