#include <hip/hip_runtime.h>

#include <cmath>
#include <chrono>
#include <cstdlib>
#include <iostream>
#include <string>
#include <fstream>
#include <assert.h>

#define PI (3.141592653589793238462643383279502884197169399375105820974944592307816406286208998628034825342117067982148086513282306647093844609550582231725359408128481)
#define TPB (1024)
#define epsilon (1e-2) //relative error
#define NUM_RUNS (5)

#define checkCUDA(status) \
{\
	if (status != hipSuccess) { \
		printf("CUDA Runtime Error: %s\n", \
			hipGetErrorString(status)); \
		assert(status == hipSuccess); \
	} \
}

typedef std::chrono::high_resolution_clock::rep hr_clock_rep;

inline hr_clock_rep get_globaltime(void)
{
	using namespace std::chrono;
	return high_resolution_clock::now().time_since_epoch().count();
}

// Returns the period in miliseconds
inline double get_timer_period(void)
{
	using namespace std::chrono;
	return 1000.0 * high_resolution_clock::period::num / high_resolution_clock::period::den;
}

hr_clock_rep timer_start, timer_stop;

/// Return true if a number is power of 2
template <typename T = unsigned>
inline bool isPowerOf2(T val)
{
    return val && (val & (val - 1)) == 0;
}

template <typename T>
inline void swap(T& x, T& y)
{
    T tmp = x; 
    x = y; 
    y = tmp; 
}

inline __device__ __host__ int LogBase2(uint64_t n)
{
    static const int table[64] = {
        0, 58, 1, 59, 47, 53, 2, 60, 39, 48, 27, 54, 33, 42, 3, 61,
        51, 37, 40, 49, 18, 28, 20, 55, 30, 34, 11, 43, 14, 22, 4, 62,
        57, 46, 52, 38, 26, 32, 41, 50, 36, 17, 19, 29, 10, 13, 21, 56,
        45, 25, 31, 35, 16, 9, 12, 44, 24, 15, 8, 23, 7, 6, 5, 63 };

    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n |= n >> 32;

    return table[(n * 0x03f6eaf2cd271461) >> 58];
}

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename TValue>
__global__ void precompute_dct_cos_kernel_backup(TValue *d_cos, TValue *scratch, int N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < N - 1)
    {
        int sum = N / 2;
        int halfLen = N / 2;
        while (tid >= sum) {
            halfLen = halfLen / 2;
            sum += halfLen;
        }
        TValue phase = (0.5 + tid - (sum - halfLen)) * PI / (halfLen << 1);
        d_cos[tid] = 0.5 / cos(phase);
    }
    else if (tid == N - 1)
    {
        d_cos[tid] = 0;
    }
}

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename TValue>
__global__ void precompute_dct_cos_kernel(TValue *d_cos, int N, int log_N)
{
    const int tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int total_height = log_N;
    if (tid < N - 1)
    {
        int k = N - tid - 1;
        // int total_height = LogBase2(N);
        int height = LogBase2(k);
        // int len = N / (1 << (total_height - height - 1));
        int len = 1 << (height + 1);
        int i = len - k - 1;
        
        TValue phase = (0.5 + i) * PI / len;
        d_cos[tid] = 0.5 / cos(phase);
    }
    else if (tid == N - 1)
    {
        d_cos[tid] = 0;
    }
}

/// Precompute cosine values needed for N-point dct
/// @param  cos  size N - 1 buffer on GPU, contains the result after function call
/// @param  N    the length of target dct, must be power of 2
template <typename TValue>
void precompute_dct_cos(TValue *cos, int N)
{
    // The input length must be power of 2
    if (! isPowerOf2<int>(N))
    {
        printf("Input length is not power of 2.\n");
        assert(0); 
    }
    timer_start = get_globaltime();

    // create the array on host 
    TValue* cos_host = new TValue [N]; 

    int offset = 0;
    int halfLen = N / 2;
    while (halfLen)
    {
        TValue phaseStep = PI / (halfLen << 1);
        // TValue phase_start = 0.5 * phaseStep;
        // #pragma omp parallel for
        for (int i = 0; i < halfLen; ++i)
        {
            TValue phase = (0.5 + i) * phaseStep;
            cos_host[offset + i] = 0.5 / std::cos(phase);
        }
        offset += halfLen;
        halfLen >>= 1;
    }
    // printf("last cos: %f\n", cos_host[N-1]);
    // while (halfLen)
    // {
    //     TValue phaseStep = 0.5 * PI / halfLen;
    //     TValue phase = 0.5 * phaseStep;
    //     for (int i = 0; i < halfLen; ++i)
    //     {
    //         cos_host[offset + i] = 0.5 / std::cos(phase);
    //         phase += phaseStep;
    //     }
    //     offset += halfLen;
    //     halfLen /= 2;
    // }

    // copy to GPU 
    hipMemcpy(cos, cos_host, N*sizeof(TValue), hipMemcpyHostToDevice);
     

    delete [] cos_host; 
    timer_stop = get_globaltime();
    printf("[D] precompute cos takes %g ms\n", (timer_stop-timer_start)*get_timer_period());
}

template <typename TValue, typename TIndex>
__global__ void computeDctForward_1(const TValue *curr_ptr, TValue *next_ptr, const TValue *cos, TIndex N, TIndex len, TIndex halfLen, TIndex cosOffset)
{
    TIndex halfN = (N >> 1);
    TIndex stride = blockDim.x * gridDim.x;
    TIndex row_id = blockIdx.y;
    const TValue * curr = curr_ptr + row_id * N;
    TValue *next = next_ptr + row_id * N;
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfN; thread_id += stride)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[offset + i] = curr[offset + i] + curr[offset + len - i - 1];
        // next[offset + i + halfLen] = (curr[offset + i] - curr[offset + len - i - 1]) * cos[cosOffset + i];
    }
    // for (TIndex thread_id = halfMN_by_gridDim*blockIdx.x + threadIdx.x; thread_id < halfMN_by_gridDim*(blockIdx.x+1); thread_id += blockDim.x)
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfN; thread_id += stride)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        //next[offset + i] = curr[offset + i] + curr[offset + len - i - 1];
        next[offset + i + halfLen] = (curr[offset + i] - curr[offset + len - i - 1]) * cos[cosOffset + i];
    }
}

template <typename TValue, typename TIndex>
__global__ void computeDctBackward_1(const TValue *curr_ptr, TValue *next_ptr, TIndex N, TIndex len, TIndex halfLen)
{
    
    TIndex halfN = (N >> 1);
    TIndex row_id = blockIdx.y;
    const TValue * curr = curr_ptr + row_id * N;
    TValue *next = next_ptr + row_id * N;
    // TIndex halfMN = M * halfN;
    //TIndex halfMN_by_gridDim = halfMN/gridDim.x;
    //for (TIndex thread_id = halfMN_by_gridDim*blockIdx.x + threadIdx.x; thread_id < halfMN_by_gridDim*(blockIdx.x+1); thread_id += blockDim.x)
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfN; thread_id += blockDim.x * gridDim.x)
    {
        TIndex rest = thread_id & (halfN - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[offset + i * 2] = curr[offset + i];
        next[offset + i * 2 + 1] = (i + 1 == halfLen) ? curr[offset + len - 1] : curr[offset + halfLen + i] + curr[offset + halfLen + i + 1];
    }
}

#define ROW2COL(IDX, COL, N) ((IDX) * (N) + (COL))

template <typename TValue, typename TIndex>
__global__ void computeDctForward_2(const TValue *curr, TValue *next, const TValue *cos, TIndex M, TIndex N, TIndex len, TIndex halfLen, TIndex cosOffset)
{
    TIndex halfM = (M >> 1);
    TIndex stride = blockDim.x * gridDim.x;
    TIndex col = blockIdx.y;
    
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfM; thread_id += stride)
    {
        TIndex rest = thread_id & (halfM - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[ROW2COL(offset + i, col, N)] = curr[ROW2COL(offset + i, col, N)] + curr[ROW2COL(offset + len - i - 1, col, N)];
        // next[offset + i + halfLen] = (curr[offset + i] - curr[offset + len - i - 1]) * cos[cosOffset + i];
    }
    // for (TIndex thread_id = halfMN_by_gridDim*blockIdx.x + threadIdx.x; thread_id < halfMN_by_gridDim*(blockIdx.x+1); thread_id += blockDim.x)
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfM; thread_id += stride)
    {
        TIndex rest = thread_id & (halfM - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        //next[offset + i] = curr[offset + i] + curr[offset + len - i - 1];
        next[ROW2COL(offset + i + halfLen, col, N)] = (curr[ROW2COL(offset + i, col, N)] - curr[ROW2COL(offset + len - i - 1, col, N)]) * cos[cosOffset + i];
    }
}

template <typename TValue, typename TIndex>
__global__ void computeDctBackward_2(const TValue *curr, TValue *next, TIndex M, TIndex N, TIndex len, TIndex halfLen)
{
    
    TIndex halfM = (M >> 1);
    TIndex col = blockIdx.y;
  
    // TIndex halfMN = M * halfN;
    //TIndex halfMN_by_gridDim = halfMN/gridDim.x;
    //for (TIndex thread_id = halfMN_by_gridDim*blockIdx.x + threadIdx.x; thread_id < halfMN_by_gridDim*(blockIdx.x+1); thread_id += blockDim.x)
    for (TIndex thread_id = blockIdx.x * blockDim.x + threadIdx.x; thread_id < halfM; thread_id += blockDim.x * gridDim.x)
    {
        TIndex rest = thread_id & (halfM - 1);
        TIndex i = rest & (halfLen - 1);
        TIndex offset = (thread_id - i) * 2;

        next[ROW2COL(offset + i * 2, col, N)] = curr[ROW2COL(offset + i, col, N)];
        next[ROW2COL(offset + i * 2 + 1, col, N)] = (i + 1 == halfLen) ? curr[ROW2COL(offset + len - 1, col, N)] : curr[ROW2COL(offset + halfLen + i, col, N)] + curr[ROW2COL(offset + halfLen + i + 1, col, N)];
    }
}

template <typename T>
__global__ void normalize(T* x, const T* y, const int M, const int N){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < M * N) {
        x[tid] = y[tid] / (M * N) * 4;
    }
}

/// The implementation of fast Discrete Cosine Transform (DCT) algorithm and its inverse (IDCT) are Lee's algorithms
/// Algorithm reference: A New Algorithm to Compute the Discrete Cosine Transform, by Byeong Gi Lee, 1984
///
/// Lee's algorithm has a recursive structure in nature.
/// Here is a sample recursive implementation: https://www.nayuki.io/page/fast-discrete-cosine-transform-algorithms
///   
/// My implementation here is iterative, which is more efficient than the recursive version.
/// Here is a sample iterative implementation: https://www.codeproject.com/Articles/151043/Iterative-Fast-1D-Forvard-DCT

/// Compute y[k] = sum_n=0..N-1 (x[n] * cos((n + 0.5) * k * PI / N)), for k = 0..N-1
/// 
/// @param  vec   length M * N sequence to be transformed in last dimension
/// @param  out   length M * N helping buffer, which is also the output
/// @param  buf   length M * N helping buffer
/// @param  cos   length N - 1, stores cosine values precomputed by function 'precompute_dct_cos'
/// @param  M     length of dimension 0 of vec  
/// @param  N     length of dimension 1 of vec, must be power of 2
template <typename TValue>
void dct_ref_1(const TValue *vec, TValue *out, TValue* buf, const TValue *cos, int M, int N)
{
    dim3 gridSize((N + TPB - 1) / TPB, M, 1);
    dim3 blockSize(TPB, 1, 1);
    // int block_count = (N + TPB - 1) / TPB; 
    // int thread_count = TPB; 

    // The input length must be power of 2
    if (! isPowerOf2<int>(N) || ! isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0); 
    }

    // Pointers point to the beginning indices of two adjacent iterations
    TValue *curr = buf; 
    TValue *next = out; 

    // 'temp' used to store date of two adjacent iterations
    // Copy 'vec' to the first N element in 'temp'
    hipMemcpy(curr, vec, M * N * sizeof(TValue), hipMemcpyDeviceToDevice);

    // Current bufferfly length and half length
    int len = N;
    int halfLen = len / 2;

    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;
    while (halfLen)
    {
        computeDctForward_1<<<gridSize, blockSize>>>(curr, next, cos, N, len, halfLen, cosOffset);
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        hipDeviceSynchronize();
        swap(curr, next);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (halfLen < N)
    {
        computeDctBackward_1<<<gridSize, blockSize>>>(curr, next, N, len, halfLen);
        halfLen = len;
        len *= 2;
        hipDeviceSynchronize();
        swap(curr, next);
        
    }

    // Populate the final results into 'out'
    // normalize<TValue><<<(N * M + TPB - 1) / TPB, TPB>>>(out, curr, M, N);
    hipDeviceSynchronize();
    
}

template <typename TValue>
void dct_ref_2(const TValue *vec, TValue *out, TValue* buf, const TValue *cos, int M, int N)
{
    dim3 gridSize((M + TPB - 1) / TPB, N, 1);
    dim3 blockSize(TPB, 1, 1);
    // int block_count = (N + TPB - 1) / TPB; 
    // int thread_count = TPB; 

    // The input length must be power of 2
    if (! isPowerOf2<int>(N) || ! isPowerOf2<int>(M))
    {
        printf("Input length is not power of 2.\n");
        assert(0); 
    }

    // Pointers point to the beginning indices of two adjacent iterations
    TValue *curr = buf; 
    TValue *next = out; 

    // 'temp' used to store date of two adjacent iterations
    // Copy 'vec' to the first N element in 'temp'
    hipMemcpy(curr, vec, M * N * sizeof(TValue), hipMemcpyDeviceToDevice);

    // Current bufferfly length and half length
    int len = M;
    int halfLen = len / 2;

    // Iteratively bi-partition sequences into sub-sequences
    int cosOffset = 0;
    while (halfLen)
    {
        computeDctForward_2<<<gridSize, blockSize>>>(curr, next, cos, M, N, len, halfLen, cosOffset);
        cosOffset += halfLen;
        len = halfLen;
        halfLen /= 2;
        hipDeviceSynchronize();
        swap(curr, next);
    }

    // Bottom-up form the final DCT solution
    // Note that the case len = 2 will do nothing, so we start from len = 4
    len = 4;
    halfLen = 2;
    while (halfLen < M)
    {
        computeDctBackward_2<<<gridSize, blockSize>>>(curr, next, M, N, len, halfLen);
        halfLen = len;
        len *= 2;
        hipDeviceSynchronize();
        swap(curr, next);
        
    }

    // Populate the final results into 'out'
    normalize<TValue><<<(N * M + TPB - 1) / TPB, TPB>>>(out, curr, M, N);
    hipDeviceSynchronize();
    
}

template <typename T>
void dct_2d_lee(
        const T *h_x,
        T *h_y,
        const int M,
        const int N
        )
{
    T *d_x;
    T *d_y;
    T *scratch;
    T *d_cos0;
    T *d_cos1;
    
    size_t size = M * N * sizeof(T);

    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_y, size);
    hipMalloc((void **)&scratch, size);
    hipMalloc((void **)&d_cos0, N * sizeof(T)); // row
    hipMalloc((void **)&d_cos1, M * sizeof(T)); // column

    hipMemcpy(d_x, h_x, size, hipMemcpyHostToDevice);
    
    hipStream_t streams[2];
    hipStreamCreate(&streams[0]);
    hipStreamCreate(&streams[1]);
    
    // precompute_dct_cos<T>(d_cos, N);
    precompute_dct_cos_kernel<T><<<(N + TPB - 1) / TPB, TPB, 0, streams[0]>>>(d_cos0, N, (int)log2(N));
    precompute_dct_cos_kernel<T><<<(M + TPB - 1) / TPB, TPB, 0, streams[1]>>>(d_cos1, M, (int)log2(M));
    hipDeviceSynchronize();

    dct_ref_1<T>(d_x, d_y, scratch, d_cos0, M, N);


    dct_ref_2<T>(d_y, d_x, scratch, d_cos1, M, N);
    
    
    hipMemcpy(h_y, d_x, size, hipMemcpyDeviceToHost);
    
    hipStreamDestroy(streams[0]);
    hipStreamDestroy(streams[1]);
    hipFree(d_x);
    hipFree(d_y);
}

template <typename T>
int validate(T* result_cuda, T* result_gt, const int N)
{
    for(int i = 0; i < N; ++i)
    {
        int flag = (std::abs(result_cuda[i] - result_gt[i]) / std::abs(result_gt[i])) < epsilon;
        if(flag == 0)
        {
            printf("%d:, cuda_res: %f, gt_res: %f\n", i, result_cuda[i], result_gt[i]);
            // return 0;
        }
    }
    return 1;
}

template <typename T>
int validate2D(T* result_cuda, T* result_gt, const int M, const int N)
{
    for(int i = 0; i < M; ++i)
    {
        for(int j = 0; j < N; ++j)
        {
            int flag = (std::abs(result_cuda[i*N+j] - result_gt[i*N+j]) / std::abs(result_gt[i*N+j])) < epsilon;
            if(flag == 0)
            {
                // printf("cuda_res[%d][%d]: %f, gt_res[%d][%d]: %f\n", i, j, result_cuda[i*N+j], i, j, result_gt[i*N+j]);
                return 0;
            }
        }
    }
    return 1;
}

template <typename T>
T** allocateMatrix(int M, int N){
    T** data;
    data = new T*[M];
    for(int i = 0; i < M; i++)
    {
        data[i] = new T[N];
    }
    return data;
}

template <typename T>
void destroyMatrix(T** &data, int M)
{
    for(int i = 0;i <M; i++){
        delete[] data[i];
    }
    delete [] data;
}

template <typename T>
void load_data(T* &data, T* &result, int &M, int &N)
{
    std::ifstream input_file("test_2d.dat", std::ios_base::in);

    int i = 0;
    T val;
    // int N;
    // int M;
    input_file >> M;
    input_file >> N;
    printf("M: %d\n", M);
    printf("N: %d\n", N);
    data = new T [M * N];
    while(input_file >> val)
    {
        data[i] = val;
        i++;
    }

    std::ifstream input_file2("result_2d.dat", std::ios_base::in);

    i = 0;
    input_file2 >> M;
    input_file2 >> N;
    result = new T [M * N];
    while(input_file2 >> val)
    {
        result[i] = val;
        i++;
    }
    printf("[I] data load done.\n");
}

typedef double dtype;
int main()
{
    dtype *h_x;
    dtype *h_y;
    dtype *h_gt;

    int M, N;
    load_data<dtype>(h_x, h_gt, M, N);
    h_y = new dtype [M * N];


    for(int i = 0;i<10;++i)
    {
        printf("%d: %f\n", i, h_x[i]);
    }
    double total_time = 0;
    for(int i = 0; i < NUM_RUNS; ++i)
    {
        timer_start = get_globaltime();
        dct_2d_lee<dtype>(h_x, h_y, M, N);
        timer_stop = get_globaltime();
        int flag = validate2D<dtype>(h_y, h_gt, M, N);
        printf("[I] validation: %d\n", flag);
        printf("[D] dct 2D takes %g ms\n", (timer_stop-timer_start)*get_timer_period());
        total_time +=  i > 0 ? (timer_stop-timer_start)*get_timer_period() : 0;
    }

    // int flag = validate<dtype>(h_y, h_gt, N);
    // printf("[D] dct 1D takes %g ms\n", (timer_stop-timer_start)*get_timer_period());
    printf("[D] dct 2D (%d * %d) takes average %g ms\n", M, N, total_time/(NUM_RUNS-1));
    // printf("[I] validation: %d\n", flag);

    for(int i = 0; i<10; ++i)
    {
        printf("%d: %f\n", i, h_y[i]);
    }

    delete [] h_x;
    delete [] h_y;
    delete [] h_gt;

    return 0;
}
