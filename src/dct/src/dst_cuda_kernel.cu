#include <stdio.h>
#include <math.h>
#include <float.h>
#include "hip/hip_runtime.h"

template <typename T>
__global__ void computeFlip(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x) 
    {
        int ii = i%N; 
        y[i] = x[i+N-ii*2-1];
    }
}

template <typename T>
void computeFlipCudaLauncher(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    computeFlip<<<32, 1024>>>(
            x, 
            M, 
            N, 
            y
            );
}

template <typename T>
__global__ void computeFlipAndShift(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*N; i += blockDim.x * gridDim.x) 
    {
        int ii = i%N; 
        y[i] = (ii)? x[i+N-ii*2] : 0;
    }
}

template <typename T>
void computeFlipAndShiftCudaLauncher(
        const T* x, 
        const int M, 
        const int N, 
        T* y
        )
{
    computeFlipAndShift<<<32, 1024>>>(
            x, 
            M, 
            N, 
            y
            );
}

template <typename T>
__global__ void negateOddEntries(
        T* x, 
        const int M, 
        const int N
        )
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < M*(N>>1); i += blockDim.x * gridDim.x) 
    {
        x[i*2+1] = -x[i*2+1]; 
    }
}

template <typename T>
void negateOddEntriesCudaLauncher(
        T* x, 
        const int M, 
        const int N
        )
{
    negateOddEntries<<<32, 1024>>>(
            x, 
            M, 
            N
            );
}

#define REGISTER_FLIP_KERNEL_LAUNCHER(type) \
    void instantiateComputeFlipLauncher(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ) \
    { \
        return computeFlipCudaLauncher<type>( \
                x, \
                M, \
                N, \
                y \
                ); \
    }

REGISTER_FLIP_KERNEL_LAUNCHER(float);
REGISTER_FLIP_KERNEL_LAUNCHER(double);

#define REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(type) \
    void instantiateComputeFlipAndShiftLauncher(\
        const type* x, \
        const int M, \
        const int N, \
        type* y \
        ) \
    { \
        return computeFlipAndShiftCudaLauncher<type>( \
                x, \
                M, \
                N, \
                y \
                ); \
    }

REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(float);
REGISTER_FLIPANDSHIFT_KERNEL_LAUNCHER(double);

#define REGISTER_NEGATE_KERNEL_LAUNCHER(type) \
    void instantiateNegateOddEntriesCudaLauncher(\
        type* x, \
        const int M, \
        const int N \
        ) \
    { \
        return negateOddEntriesCudaLauncher<type>( \
                x, \
                M, \
                N \
                ); \
    }

REGISTER_NEGATE_KERNEL_LAUNCHER(float);
REGISTER_NEGATE_KERNEL_LAUNCHER(double);
